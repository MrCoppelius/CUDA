#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


#define ErrorCheck(ans) { CheckFun((ans), __FILE__, __LINE__); }

inline void CheckFun(hipError_t code, const char *file, int line){
   if (code != hipSuccess) {
      fprintf(stderr,"ERROR: %s %s %d\n", hipGetErrorString(code), file, line);
      exit(0);
   }
}

__global__ void multiply(double* dev_A, double* dev_B, size_t arrLen) {
    size_t index = threadIdx.x + blockIdx.x * blockDim.x;
    while (index < arrLen) {
        dev_A[index] = dev_A[index] * dev_B[index];
        index += blockDim.x * gridDim.x;
    }
}


int main() {
    size_t arrLen;
    scanf("%zd", &arrLen);
    size_t size = sizeof(double) * arrLen;
    double *arrA = (double*)malloc(size);
    double *arrB = (double*)malloc(size);
    for (size_t i = 0; i < arrLen; ++i) {
        scanf("%lf", &arrA[i]);
    }
    for (size_t i = 0; i < arrLen; ++i) {
        scanf("%lf", &arrB[i]);
    }
    double *dev_A, *dev_B;
    ErrorCheck(hipMalloc((void**)&dev_A, size));
    ErrorCheck(hipMalloc((void**)&dev_B, size));
    ErrorCheck(hipMemcpy(dev_A, arrA, size, hipMemcpyHostToDevice));
    ErrorCheck(hipMemcpy(dev_B, arrB, size, hipMemcpyHostToDevice));
    dim3 blockSize = dim3(512,1,1);
    dim3 gridSize = dim3((unsigned int)arrLen / 512 + 1, 1, 1);
    multiply <<<gridSize, blockSize >>> (dev_A, dev_B, arrLen);
    ErrorCheck(hipGetLastError());
    ErrorCheck(hipMemcpy(arrA, dev_A, size, hipMemcpyDeviceToHost));
    for (size_t i = 0; i < arrLen; ++i) {
        printf("%.10lf ", arrA[i]);
    }
    printf("\n");
    free(arrA);
    free(arrB);
    ErrorCheck(hipFree(dev_A));
    ErrorCheck(hipFree(dev_B));

    return 0;
 }
